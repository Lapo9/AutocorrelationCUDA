#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include "Feeder.h"
#include "DataFile.h"
#include "CudaWindow.h"
#include "CudaInput.h"
#include "InputVector.h"
#include <iostream>
#include <chrono>
#include <thread>
#include <vector>
#include <memory>

#define BLOCK_SIZE 100
#define MAX_LAG 60
#define THREADS_PER_BLOCK 32


template <int maxLag, int blockSize, typename Contained>
__global__ void autocorrelate(AutocorrelationCUDA::CudaWindow<maxLag, blockSize, Contained> window, int start, int* out);


int main() {

	//open file where data is stored
	std::unique_ptr<AutocorrelationCUDA::CudaInput<std::uint8_t>> dataFile = std::make_unique<AutocorrelationCUDA::InputVector<std::uint8_t>>("C:\\", "test1");
	
	//array in GPU memory to store output data
	int* out_d;
	hipMalloc(&out_d, MAX_LAG * sizeof(int));

	//create circular array in GPU memory
	AutocorrelationCUDA::CudaWindow<MAX_LAG, BLOCK_SIZE, std::uint8_t> window{};

	int timesCalled; //counter
	dim3 numberOfBlocks = ceil((float)MAX_LAG / THREADS_PER_BLOCK); //number of blocks active on the GPU
	
	//timer
	AutocorrelationCUDA::Timer timer{[](std::vector<double> data){AutocorrelationCUDA::DataFile<double>::write(data, "timer_out.txt");},
									 [](){return 1.234;}};

	for(timesCalled = 0; timesCalled < 16; ++timesCalled) {
		timer.start();
		window.copyBlock(dataFile->read(BLOCK_SIZE), hipMemcpyHostToDevice); //store in GPU memory one block of data
		autocorrelate <<< numberOfBlocks, THREADS_PER_BLOCK >>> (window, timesCalled * BLOCK_SIZE, out_d);
		hipDeviceSynchronize();
		timer.getInterval();
	}

	//copy output data from GPU to CPU
	std::vector<int> out(MAX_LAG);
	hipMemcpy(out.data(), out_d, MAX_LAG * sizeof(int), hipMemcpyDeviceToHost);

	window.clean(); //deallocates memory on GPU

	std::cout << timesCalled << "\n";
	for (int i = 0; i < MAX_LAG; ++i) {
		out[i] = out[i] / ((timesCalled * BLOCK_SIZE) - i);
		std::cout << i << " --> " << out[i] << std::endl;
	}

	//write output to file
	AutocorrelationCUDA::DataFile<int>::write(out);


	

}


template <int maxLag, int blockSize, typename Contained>
__global__ void autocorrelate(AutocorrelationCUDA::CudaWindow<maxLag, blockSize, Contained> window, int start, int* out) {
	if(threadIdx.x <= MAX_LAG){
		int absoluteThreadsIdx = blockIdx.x * blockDim.x + threadIdx.x;
		int partialSum = 0;
		for (int i = 0; i < BLOCK_SIZE; ++i) {
			if(i+start - absoluteThreadsIdx >= 0) {
				int a = window[i + start - absoluteThreadsIdx];
				int b = window[i + start];
				partialSum += a*b;
				//partialSum += window[i+start - threadIdx.x] * window[i+start];
			}
		}
		
		out[absoluteThreadsIdx] += partialSum;
	}
}
