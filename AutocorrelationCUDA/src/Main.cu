#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include "Feeder.h"
#include "DataFile.h"
#include "CudaWindow.h"
#include <iostream>
#include <chrono>
#include <thread>
#include <vector>

#define BLOCK_SIZE 4
#define MAX_LAG 10

using namespace std::chrono_literals;

__global__ void autocorrelate(AutocorrelationCUDA::CudaWindow<MAX_LAG, BLOCK_SIZE, std::uint8_t> window, int start, int* out);


int main() {
	
	//read file where data is stored
	AutocorrelationCUDA::DataFile<std::uint8_t> dataFile{"C:\\", "test1"};
	
	//copy read data to GPU
	AutocorrelationCUDA::CudaWindow<MAX_LAG, BLOCK_SIZE, std::uint8_t> window{};
	window.copyBlock(dataFile.read(BLOCK_SIZE), hipMemcpyHostToDevice);

	//array in memory to store output data
	int* out_d;
	hipMalloc(&out_d, BLOCK_SIZE * sizeof(int));

	autocorrelate <<< 1, MAX_LAG >>> (window, 0, out_d);

	//copy output data from GPU to CPU
	std::vector<int> out(BLOCK_SIZE);
	hipMemcpy(out.data(), out_d, BLOCK_SIZE * sizeof(int), hipMemcpyDeviceToHost);

	//write output to file
	dataFile.write<int>(out);

	for (int i = 0; i < BLOCK_SIZE; ++i) {
		std::cout << out[i] << std::endl;
	}

	/*Feeder f1{2s, [] {std::cout << "Ciao!\n" << std::endl; }};
	f1.start();

	std::this_thread::sleep_for(4s);
	f1.pause();
	std::this_thread::sleep_for(6s);
	f1.resume();

	std::this_thread::sleep_for(10s); */
	//receive data
	//send data to GPU
	//launch kernel
	//loop

	//collect results

}


__global__ void autocorrelate(AutocorrelationCUDA::CudaWindow<MAX_LAG, BLOCK_SIZE, std::uint8_t> window, int start, int* out) {

	int partialSum = 0;
	if(threadIdx.x <= MAX_LAG){
		for (int i = 0; i < BLOCK_SIZE; ++i) {
			if(threadIdx.x+i < BLOCK_SIZE){
				partialSum += window[i] * window[threadIdx.x + i];
			}
		}
		out[threadIdx.x] = partialSum;
	}

}